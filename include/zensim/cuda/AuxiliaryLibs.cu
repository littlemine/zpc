#include "AuxiliaryLibs.cuh"

namespace zs {

  CuBlas::CuBlas() {
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&handle);
    checkCudaErrors(hipblasStatus_t);
  }
  CuBlas::~CuBlas() {
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasDestroy(handle);
    checkCudaErrors(hipblasStatus_t);
  }
  CuSparse::CuSparse() {
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&handle);
    checkCudaErrors(cusparseStatus);
    hipsparseSetStream(handle, (hipStream_t)Cuda::ref_cuda_context(0).streamCompute());
  }
  CuSparse::~CuSparse() {
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseDestroy(handle);
    checkCudaErrors(cusparseStatus);
  }

  CuSolverSp::CuSolverSp() {
    hipsolverStatus_t cusolverStatus;
    cusolverStatus = hipsolverSpCreate(&handle);
    checkCudaErrors(cusolverStatus);
    hipsolverSpSetStream(handle, (hipStream_t)Cuda::ref_cuda_context(0).streamCompute());
  }
  CuSolverSp::~CuSolverSp() {
    hipsolverStatus_t cusolverStatus;
    cusolverStatus = hipsolverSpDestroy(handle);
    checkCudaErrors(cusolverStatus);
  }
  CuSolverDn::CuSolverDn() {
    hipsolverStatus_t cusolverStatus;
    cusolverStatus = hipsolverDnCreate(&handle);
    checkCudaErrors(cusolverStatus);
    hipsolverSetStream(handle, (hipStream_t)Cuda::ref_cuda_context(0).streamCompute());
  }
  CuSolverDn::~CuSolverDn() {
    hipsolverStatus_t cusolverStatus;
    cusolverStatus = hipsolverDnDestroy(handle);
    checkCudaErrors(cusolverStatus);
  }

}  // namespace zs