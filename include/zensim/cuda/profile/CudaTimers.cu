#include "CudaTimers.cuh"
#include "zensim/zpc_tpls/fmt/color.h"
#include "zensim/zpc_tpls/fmt/core.h"

namespace zs {

  CudaTimer::CudaTimer(stream_t sid) : streamId{sid} {
    hipEventCreateWithFlags((hipEvent_t *)&last, hipEventBlockingSync);
    hipEventCreateWithFlags((hipEvent_t *)&cur, hipEventBlockingSync);
  }
  CudaTimer::~CudaTimer() {
    hipEventDestroy((hipEvent_t)last);
    hipEventDestroy((hipEvent_t)cur);
  }
  float CudaTimer::elapsed() {
    float duration;
    hipEventSynchronize((hipEvent_t)cur);
    hipEventElapsedTime(&duration, (hipEvent_t)last, (hipEvent_t)cur);
    return duration;
  }
  void CudaTimer::tick() { hipEventRecord((hipEvent_t)last, (hipStream_t)streamId); }
  void CudaTimer::tock() { hipEventRecord((hipEvent_t)cur, (hipStream_t)streamId); }
  void CudaTimer::tock(std::string_view tag) {
    tock();
    fmt::print(fg(fmt::color::cyan), "{}: {} ms\n", tag, elapsed());
  }

}  // namespace zs