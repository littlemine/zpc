#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <utility>

#include "../Platform.hpp"
#include "Cuda.h"
#include "zensim/zpc_tpls/fmt/color.h"
#include "zensim/zpc_tpls/fmt/format.h"
#include "zensim/types/SourceLocation.hpp"

#define MEM_POOL_CTRL 3

namespace zs {

  /*
    __device__ __constant__ char g_cuda_constant_cache[8192];  // 1024 words

    void Cuda::init_constant_cache(void *ptr, std::size_t size) {
      hipMemcpyToSymbol(HIP_SYMBOL(g_cuda_constant_cache), ptr, size, 0, hipMemcpyHostToDevice);
      // cudri::memcpyHtoD((void *)g_cuda_constant_cache, ptr, size);
    }
    void Cuda::init_constant_cache(void *ptr, std::size_t size, void *stream) {
      hipMemcpyToSymbolAsync(HIP_SYMBOL(g_cuda_constant_cache), ptr, size, 0, hipMemcpyHostToDevice,
                              (hipStream_t)stream);
    }
    */

  /// error handling
  u32 Cuda::get_last_cuda_rt_error() { return (u32)hipGetLastError(); }

  std::string_view Cuda::get_cuda_rt_error_string(u32 errorCode) {
    // return hipGetErrorString((hipError_t)errorCode);
    return hipGetErrorString((hipError_t)errorCode);
  }
  void Cuda::check_cuda_rt_error(u32 errorCode, ProcID did, const source_location &loc) {
    if (errorCode != 0) {
      if (did >= 0) {
        auto &context = Cuda::context(did);
        if (context.errorStatus) return;  // there already exists a preceding cuda error
        context.errorStatus = true;
      }
      const auto fileInfo = fmt::format("# File: \"{:<50}\"", loc.file_name());
      const auto locInfo = fmt::format("# Ln {}, Col {}", loc.line(), loc.column());
      const auto funcInfo = fmt::format("# Func: \"{}\"", loc.function_name());
      fmt::print(fg(fmt::color::crimson) | fmt::emphasis::italic | fmt::emphasis::bold,
                 "\nCuda Error on Device {}: {}\n{:=^60}\n{}\n{}\n{}\n{:=^60}\n\n",
                 did >= 0 ? std::to_string(did) : "unknown", get_cuda_rt_error_string(errorCode),
                 " cuda api error location ", fileInfo, locInfo, funcInfo, "=");
    }
  }

  /// kernel launch
  u32 Cuda::launchKernel(const void *f, unsigned int gx, unsigned int gy, unsigned int gz,
                         unsigned int bx, unsigned int by, unsigned int bz, void **args,
                         std::size_t shmem, void *stream) {
    return hipLaunchKernel(reinterpret_cast<const void*>(f), dim3{gx, gy, gz}, dim3{bx, by, bz}, args, shmem,
                            (hipStream_t)stream);
    // return cudri::launchCuKernel(const_cast<void *>(f), gx, gy, gz, bx, by, bz, (unsigned
    // int)shmem,
    //                      stream, args, (void **)nullptr);
  }
  u32 Cuda::launchCooperativeKernel(const void *f, unsigned int gx, unsigned int gy,
                                    unsigned int gz, unsigned int bx, unsigned int by,
                                    unsigned int bz, void **args, std::size_t shmem, void *stream) {
    // return cudri::launchCuCooperativeKernel(const_cast<void *>(f), gx, gy, gz, bx, by, bz, shmem,
    // stream, args);
    return hipLaunchCooperativeKernel(reinterpret_cast<const void*>(f), dim3{gx, gy, gz}, dim3{bx, by, bz}, args, shmem,
                                       (hipStream_t)stream);
  }
  u32 Cuda::launchCallback(void *stream, void *f, void *data) {
    // return hipLaunchHostFunc((hipStream_t)stream, (hipHostFn_t)f, data);
    return (u32)cudri::launchHostFunc(stream, f, data);
  }

  void Cuda::CudaContext::checkError(u32 errorCode, const source_location &loc) const {
    /// only shows the first error message
    Cuda::check_cuda_rt_error(errorCode, getDevId(), loc);
  }

  // record
  void Cuda::CudaContext::recordEventCompute(const source_location &loc) {
    // checkError(hipEventRecord((hipEvent_t)eventCompute(), (hipStream_t)streamCompute()), loc);
    cudri::recordEvent(eventCompute(), streamCompute());
  }
  void Cuda::CudaContext::recordEventSpare(unsigned id, const source_location &loc) {
    // checkError(hipEventRecord((hipEvent_t)eventSpare(id), (hipStream_t)streamSpare(id)), loc);
    cudri::recordEvent(eventSpare(id), streamSpare(id));
  }
  // sync
  void Cuda::CudaContext::syncStream(unsigned sid, const source_location &loc) const {
    // checkError(hipStreamSynchronize((hipStream_t)stream(sid)), loc);
    cudri::syncStream(stream(sid));
  }
  void Cuda::CudaContext::syncCompute(const source_location &loc) const {
    // checkError(hipStreamSynchronize((hipStream_t)streamCompute()), loc);
    cudri::syncStream(streamCompute());
  }
  void Cuda::CudaContext::syncStreamSpare(unsigned sid, const source_location &loc) const {
    // checkError(hipStreamSynchronize((hipStream_t)streamSpare(sid)), loc);
    cudri::syncStream(streamSpare(sid));
  }
  // stream-event sync
  void Cuda::CudaContext::computeStreamWaitForEvent(void *event, const source_location &loc) {
    cudri::streamWaitEvent(streamCompute(), event, 0);
  }
  void Cuda::CudaContext::spareStreamWaitForEvent(unsigned sid, void *event,
                                                  const source_location &loc) {
    // checkError(hipStreamWaitEvent((hipStream_t)streamSpare(sid), (hipEvent_t)event, 0),
    // loc);
    cudri::streamWaitEvent(streamSpare(sid), event, 0);
  }
  void *Cuda::CudaContext::streamMemAlloc(std::size_t size, void *stream,
                                          const source_location &loc) {
    void *ptr;
    cudri::mallocAsync(&ptr, size, stream);
    return ptr;
  }
  void Cuda::CudaContext::streamMemFree(void *ptr, void *stream, const source_location &loc) {
    cudri::freeAsync(ptr, stream);
  }
  Cuda::CudaContext::StreamExecutionTimer *Cuda::CudaContext::tick(void *stream,
                                                                   const source_location &loc) {
    return new StreamExecutionTimer(this, stream, loc);
  }
  void Cuda::CudaContext::tock(Cuda::CudaContext::StreamExecutionTimer *timer,
                               const source_location &loc) {
    cudri::launchHostFunc(timer->stream, (void *)recycle_timer, (void *)timer);
  }

  void Cuda::CudaContext::setContext(const source_location &loc) const {
    cudri::setContext{getContext()};
  }

  namespace cudri {

    void load_cuda_driver_apis() {
      auto initializer = []() -> bool {
        hipInit(0);
        // int trialVer = CUDA_VERSION - CUDA_VERSION % 1000;
        int trialVer = 11030;
        while (trialVer > 0
               && hipGetProcAddress("hipDrvGetErrorName", (void **)&get_cu_error_name, trialVer,
                                   hipEnableDefault)
                      == hipErrorNotFound)
          trialVer -= 1000;
        const int ver = trialVer;
        fmt::print(fg(fmt::color::green_yellow) | fmt::emphasis::underline,
                   "loading cuda driver api version: {}\n", ver);
        hipGetProcAddress("hipDrvGetErrorString", (void **)&get_cu_error_string, ver,
                         hipEnableDefault);

#define PER_CUDA_FUNCTION(name, symbol_name, ...)                        \
  hipGetProcAddress(#symbol_name, (void **)&name<__VA_ARGS__>::func, ver, \
                   hipEnableDefault);
#include "cuda_driver_functions.inc.h"
#undef PER_CUDA_FUNCTION
        return true;
      };
      static bool initialized = initializer();
    }
    // template struct name<__VA_ARGS__>;
  }  // namespace cudri

  Cuda::Cuda() {
    fmt::print("[Init -- Begin] Cuda\n");
    errorStatus = false;
    cudri::load_cuda_driver_apis();

    numTotalDevice = 0;
    cudri::getDeviceCount{&numTotalDevice};
    contexts.resize(numTotalDevice);
    if (numTotalDevice == 0)
      fmt::print(
          "\t[InitInfo -- DevNum] There are no available device(s) that "
          "support CUDA\n");
    else
      fmt::print("\t[InitInfo -- DevNum] Detected {} CUDA Capable device(s)\n", numTotalDevice);

    for (int i = 0; i < numTotalDevice; i++) {
      auto &context = contexts[i];
      int dev{};
      {
        void *c{nullptr};
        // checkError(hipSetDevice(i), i);
        cudri::getDevice(&dev, i);
        // fmt::print("device ordinal {} is {}\n", i, dev);

        // getContext(&c);
        // cudri::retainDevicePrimaryCtx(&c, dev);
        cudri::createContext(&c, 0,
                             dev);  // hipDeviceScheduleBlockingSync(0x04) | hipDeviceScheduleSpin(0x01)
        context = CudaContext{i, dev, c};
      }

      context.streams.resize((int)StreamIndex::Total);
      for (auto &stream : context.streams)
        // checkError(hipStreamCreateWithFlags((hipStream_t *)&stream, hipStreamNonBlocking),
        // i);
        cudri::createStream(&stream, 0x01);
      context.events.resize((int)EventIndex::Total);
      for (auto &event : context.events)
        // checkError(hipEventCreateWithFlags((hipEvent_t *)&event, hipEventBlockingSync), i);
        cudri::createEvent(&event, 0x01);

      {  ///< device properties
        using namespace cudri;
        int major, minor, multiGpuBoardGroupID, multiProcessorCount, regsPerBlock;
        int supportUnifiedAddressing, supportUm, supportConcurrentUmAccess;
        getDeviceAttribute(&regsPerBlock, (unsigned)hipDeviceAttributeMaxRegistersPerBlock,
                           dev);
        getDeviceAttribute(&multiProcessorCount, (unsigned)hipDeviceAttributeMultiprocessorCount,
                           dev);
        getDeviceAttribute(&multiGpuBoardGroupID,
                           (unsigned)hipDeviceAttributeMultiGpuBoardGroupId, dev);
        getDeviceAttribute(&textureAlignment, (unsigned)hipDeviceAttributeTextureAlignment, dev);
        getDeviceAttribute(&minor, (unsigned)hipDeviceAttributeComputeCapabilityMinor, dev);
        getDeviceAttribute(&major, (unsigned)hipDeviceAttributeComputeCapabilityMajor, dev);
        getDeviceAttribute(&supportUnifiedAddressing,
                           (unsigned)hipDeviceAttributeUnifiedAddressing, dev);
        getDeviceAttribute(&supportUm, (unsigned)hipDeviceAttributeManagedMemory, dev);
        getDeviceAttribute(&supportConcurrentUmAccess,
                           (unsigned)hipDeviceAttributeConcurrentManagedAccess, dev);
        getDeviceAttribute(&context.regsPerMultiprocessor,
                           (unsigned)hipDeviceAttributeMaxRegistersPerMultiprocessor, dev);
        getDeviceAttribute(&context.sharedMemPerMultiprocessor,
                           (unsigned)hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, dev);
        getDeviceAttribute(&context.maxBlocksPerMultiprocessor,
                           (unsigned)hipDeviceAttributeMaxBlocksPerMultiprocessor, dev);
        getDeviceAttribute(&context.sharedMemPerBlock,
                           (unsigned)hipDeviceAttributeMaxSharedMemoryPerBlock, dev);
        getDeviceAttribute(&context.maxThreadsPerBlock,
                           (unsigned)hipDeviceAttributeMaxThreadsPerBlock, dev);
        getDeviceAttribute(&context.maxThreadsPerMultiprocessor,
                           (unsigned)hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);

        context.supportConcurrentUmAccess = supportConcurrentUmAccess;

        fmt::print(
            "\t[InitInfo -- Dev Property] GPU device {} ({}-th group on "
            "board)\n\t\tshared memory per block: {} bytes,\n\t\tregisters per SM: "
            "{},\n\t\tMulti-Processor count: {},\n\t\tSM compute capabilities: "
            "{}.{}.\n\t\tTexture alignment: {} bytes\n\t\tUVM support: allocation({}), unified "
            "addressing({}), concurrent access({})\n",
            i, multiGpuBoardGroupID, context.sharedMemPerBlock, regsPerBlock, multiProcessorCount,
            major, minor, textureAlignment, supportUm, supportUnifiedAddressing,
            supportConcurrentUmAccess);
      }
    }

    /// enable peer access if feasible
    for (int i = 0; i < numTotalDevice; i++) {
      // checkError(hipSetDevice(i), i);
      cudri::setContext(contexts[i].getContext());
      for (int j = 0; j < numTotalDevice; j++) {
        if (i != j) {
          int iCanAccessPeer = 0;
          cudri::canAccessPeer(&iCanAccessPeer, contexts[i].getDevice(), contexts[j].getDevice());
          if (iCanAccessPeer) cudri::enablePeerAccess(contexts[j].getContext(), 0);
          fmt::print("\t[InitInfo -- Peer Access] Peer access status {} -> {}: {}\n", i, j,
                     iCanAccessPeer ? "Inactive" : "Active");
        }
      }
    }
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#um-requirements
    /* GPUs with SM architecture 6.x or higher (Pascal class or newer) provide additional
    Unified Memory features such as on-demand page migration and GPU memory oversubscription
    that are outlined throughout this document. Note that currently these features are only
    supported on Linux operating systems. Applications running on Windows (whether in TCC
    or WDDM mode) will use the basic Unified Memory model as on pre-6.x architectures even
    when they are running on hardware with compute capability 6.x or higher. */

    fmt::print("\n[Init -- End] == Finished \'Cuda\' initialization\n\n");
  }

  Cuda::~Cuda() {
#if 0
    /// let driver automatically recycles resource 
    for (int i = 0; i < numTotalDevice; i++) {
      auto &context = contexts[i];
      context.setContext();
      // checkError(hipDeviceSynchronize(), i);
      cudri::syncContext();
      for (auto stream : context.streams)
        cudri::destroyStream{stream};  // checkError(hipStreamDestroy((hipStream_t)stream), i);
      for (auto event : context.events)
        cudri::destroyEvent{event};  // checkError(hipEventDestroy((hipEvent_t)event), i);
      context.deviceMem.reset(nullptr);
      context.unifiedMem.reset(nullptr);

      // destroyContext(context.getContext());
      // checkError(hipDeviceReset(), i);
      cudri::destroyContext(context.getContext());
    }
    fmt::print("  Finished \'Cuda\' termination\n");
#endif
  }

  /// reference: kokkos/core/src/Cuda/Kokkos_Cuda_BlockSize_Deduction.hpp, Ln 101
  int Cuda::deduce_block_size(const Cuda::CudaContext &ctx, void *kernelFunc,
                              std::function<std::size_t(int)> block_size_to_dynamic_shmem,
                              std::string_view kernelName) {
    if (auto it = ctx.funcLaunchConfigs.find(kernelFunc); it != ctx.funcLaunchConfigs.end())
      return it->second.optBlockSize;
    hipFuncAttributes funcAttribs;
    ctx.checkError(hipFuncGetAttributes(&funcAttribs, reinterpret_cast<const void*>(kernelFunc)));
    int optBlockSize{0};

    auto cuda_max_active_blocks_per_sm = [&](int block_size, int dynamic_shmem) {
      // Limits due do registers/SM
      int const regs_per_sm = ctx.regsPerMultiprocessor;
      int const regs_per_thread = funcAttribs.numRegs;
      int const max_blocks_regs = regs_per_sm / (regs_per_thread * block_size);

      // Limits due to shared memory/SM
      size_t const shmem_per_sm = ctx.sharedMemPerMultiprocessor;
      size_t const shmem_per_block = ctx.sharedMemPerBlock;
      size_t const static_shmem = funcAttribs.sharedSizeBytes;
      size_t const dynamic_shmem_per_block = funcAttribs.maxDynamicSharedSizeBytes;
      size_t const total_shmem = static_shmem + dynamic_shmem;

      int const max_blocks_shmem
          = total_shmem > shmem_per_block || dynamic_shmem > dynamic_shmem_per_block
                ? 0
                : (total_shmem > 0 ? (int)shmem_per_sm / total_shmem : max_blocks_regs);

      // Limits due to blocks/SM
      int const max_blocks_per_sm = ctx.maxBlocksPerMultiprocessor;

      // Overall occupancy in blocks
      return std::min({max_blocks_regs, max_blocks_shmem, max_blocks_per_sm});
    };
    auto deduce_opt_block_size = [&]() {
      // Limits
      int const max_threads_per_sm = ctx.maxThreadsPerMultiprocessor;
      // unsure if I need to do that or if this is already accounted for in the functor attributes
      int const min_blocks_per_sm = 1;
      int const max_threads_per_block
          = std::min((int)ctx.maxThreadsPerBlock, funcAttribs.maxThreadsPerBlock);

      // Recorded maximum
      int opt_block_size = 0;
      int opt_threads_per_sm = 0;

      /// iterate all optional blocksize setup
      for (int block_size = max_threads_per_block; block_size > 0; block_size -= 32) {
        size_t const dynamic_shmem = block_size_to_dynamic_shmem(block_size);

        int blocks_per_sm = cuda_max_active_blocks_per_sm(block_size, dynamic_shmem);

        int threads_per_sm = blocks_per_sm * block_size;

        if (threads_per_sm > max_threads_per_sm) {
          blocks_per_sm = max_threads_per_sm / block_size;
          threads_per_sm = blocks_per_sm * block_size;
        }

        // update if higher occupancy (more threads per streaming multiprocessor)
        if (blocks_per_sm >= min_blocks_per_sm) {
          if (threads_per_sm >= opt_threads_per_sm) {
            opt_block_size = block_size;
            opt_threads_per_sm = threads_per_sm;
          }
        }

        // fmt::print("current blocks_sm: {}, threads_sm: {}, size: {}\n", blocks_per_sm,
        //           threads_per_sm, block_size);
        // if (blocks_per_sm != 0) break; // this enabled when querying for maximum block size
      }
      return opt_block_size;
    };

    optBlockSize = deduce_opt_block_size();
    fmt::print(
        fg(fmt::color::lime_green) | fmt::emphasis::bold,
        "{:=^60}\nnumRegs: {}\t\tmaxThreadsPerBlock: {}\nsharedSizeBytes: {}\t"
        "maxDynamicSharedSizeBytes: {}.\n",
        fmt::format(" cuda kernel [{}] optBlockSize [{}] ",
                    kernelName.empty() ? std::to_string((std::uintptr_t)kernelFunc) : kernelName,
                    optBlockSize),
        funcAttribs.numRegs, funcAttribs.maxThreadsPerBlock, funcAttribs.sharedSizeBytes,
        funcAttribs.maxDynamicSharedSizeBytes);
    ctx.funcLaunchConfigs.emplace(kernelFunc, typename Cuda::CudaContext::Config{optBlockSize});
    return optBlockSize;
  }

}  // namespace zs
