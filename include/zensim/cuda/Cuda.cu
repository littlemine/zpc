#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <utility>

#include "../Platform.hpp"
#include "Cuda.h"
#include "zensim/tpls/fmt/format.h"
#include "zensim/types/SourceLocation.hpp"

#define MEM_POOL_CTRL 3

namespace zs {

  __device__ __constant__ char g_cuda_constant_cache[8192];  // 1024 words

  void Cuda::init_constant_cache(void *ptr, std::size_t size) {
    hipMemcpyToSymbol(HIP_SYMBOL(g_cuda_constant_cache), ptr, size, 0, hipMemcpyHostToDevice);
    // cudri::memcpyHtoD((void *)g_cuda_constant_cache, ptr, size);
  }
  void Cuda::init_constant_cache(void *ptr, std::size_t size, void *stream) {
    hipMemcpyToSymbolAsync(HIP_SYMBOL(g_cuda_constant_cache), ptr, size, 0, hipMemcpyHostToDevice,
                            (hipStream_t)stream);
  }

  std::string get_cu_error_message(u32 err) {
    const char *err_name_ptr;
    const char *err_string_ptr;
    cudri::get_cu_error_name(err, &err_name_ptr);
    cudri::get_cu_error_string(err, &err_string_ptr);
    return fmt::format("CUDA Driver Error {}: {}", err_name_ptr, err_string_ptr);
  }

  /// error handling
  u32 Cuda::getLastCudaError() { return (u32)hipGetLastError(); }

  std::string_view Cuda::getCudaErrorString(u32 errorCode) {
    // return hipGetErrorString((hipError_t)errorCode);
    return hipGetErrorString((hipError_t)errorCode);
  }
  void Cuda::checkError(u32 errorCode, ProcID did, const source_location &loc) {
    if (errorCode != 0) {
      if (did >= 0) {
        auto &context = Cuda::context(did);
        if (context.errorStatus) return;  // there already exists a preceding cuda error
        context.errorStatus = true;
      }
      const auto fileInfo = fmt::format("# File: \"{}\"", loc.file_name());
      const auto locInfo = fmt::format("# Ln {}, Col {}", loc.line(), loc.column());
      const auto funcInfo = fmt::format("# Func: \"{}\"", loc.function_name());
      fmt::print(fg(fmt::color::crimson) | fmt::emphasis::italic | fmt::emphasis::bold,
                 "\nCuda Error on Device {}: {}\n{:=^60}\n{}\n{}\n{}\n{:=^60}\n\n",
                 did >= 0 ? std::to_string(did) : "unknown", get_cu_error_message(errorCode),
                 " cuda api error location ", fileInfo, locInfo, funcInfo, "=");
    }
  }

  /// kernel launch
  u32 Cuda::launchKernel(const void *f, unsigned int gx, unsigned int gy, unsigned int gz,
                         unsigned int bx, unsigned int by, unsigned int bz, void **args,
                         std::size_t shmem, void *stream) {
    return hipLaunchKernel(reinterpret_cast<const void*>(f), dim3{gx, gy, gz}, dim3{bx, by, bz}, args, shmem,
                            (hipStream_t)stream);
    // cudri::launchCuKernel(const_cast<void *>(f), gx, gy, gz, bx, by, bz, (unsigned int)shmem,
    //                      stream, args, (void **)nullptr);
    // return 0;
  }
  u32 Cuda::launchCooperativeKernel(const void *f, unsigned int gx, unsigned int gy,
                                    unsigned int gz, unsigned int bx, unsigned int by,
                                    unsigned int bz, void **args, std::size_t shmem, void *stream) {
    // cudri::launchCuCooperativeKernel(const_cast<void *>(f), gx, gy, gz, bx, by, bz, shmem,
    // stream,
    //                                 args);
    // return 0;
    return hipLaunchCooperativeKernel(reinterpret_cast<const void*>(f), dim3{gx, gy, gz}, dim3{bx, by, bz}, args, shmem,
                                       (hipStream_t)stream);
  }
  u32 Cuda::launchCallback(void *stream, void *f, void *data) {
    // return hipLaunchHostFunc((hipStream_t)stream, (hipHostFn_t)f, data);
    return (u32)cudri::launchHostFunc(stream, f, data);
  }

  void Cuda::CudaContext::checkError() const {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
      fmt::print("Last Error on [Dev {}]: {}\n", devid, hipGetErrorString(error));
  }

  void Cuda::CudaContext::checkError(u32 errorCode, const source_location &loc) const {
    /// only shows the first error message
    Cuda::checkError(errorCode, getDevId(), loc);
  }

  // record
  void Cuda::CudaContext::recordEventCompute(const source_location &loc) {
    // checkError(hipEventRecord((hipEvent_t)eventCompute(), (hipStream_t)streamCompute()), loc);
    cudri::recordEvent(eventCompute(), streamCompute());
  }
  void Cuda::CudaContext::recordEventSpare(unsigned id, const source_location &loc) {
    // checkError(hipEventRecord((hipEvent_t)eventSpare(id), (hipStream_t)streamSpare(id)), loc);
    cudri::recordEvent(eventSpare(id), streamSpare(id));
  }
  // sync
  void Cuda::CudaContext::syncStream(unsigned sid, const source_location &loc) const {
    // checkError(hipStreamSynchronize((hipStream_t)stream(sid)), loc);
    cudri::syncStream(stream(sid));
  }
  void Cuda::CudaContext::syncCompute(const source_location &loc) const {
    // checkError(hipStreamSynchronize((hipStream_t)streamCompute()), loc);
    cudri::syncStream(streamCompute());
  }
  void Cuda::CudaContext::syncStreamSpare(unsigned sid, const source_location &loc) const {
    // checkError(hipStreamSynchronize((hipStream_t)streamSpare(sid)), loc);
    cudri::syncStream(streamSpare(sid));
  }
  // stream-event sync
  void Cuda::CudaContext::computeStreamWaitForEvent(void *event, const source_location &loc) {
    cudri::streamWaitEvent(streamCompute(), event, 0);
  }
  void Cuda::CudaContext::spareStreamWaitForEvent(unsigned sid, void *event,
                                                  const source_location &loc) {
    // checkError(hipStreamWaitEvent((hipStream_t)streamSpare(sid), (hipEvent_t)event, 0),
    // loc);
    cudri::streamWaitEvent(streamSpare(sid), event, 0);
  }
  void *Cuda::CudaContext::streamMemAlloc(std::size_t size, void *stream,
                                          const source_location &loc) {
    void *ptr;
    // checkError(hipMallocAsync(&ptr, size, (hipStream_t)stream), loc);
    cudri::mallocAsync(&ptr, size, stream);
    return ptr;
  }
  void Cuda::CudaContext::streamMemFree(void *ptr, void *stream, const source_location &loc) {
    // checkError(hipFreeAsync(ptr, (hipStream_t)stream), loc);
    cudri::freeAsync(ptr, stream);
  }
  Cuda::CudaContext::StreamExecutionTimer *Cuda::CudaContext::tick(void *stream,
                                                                   const source_location &loc) {
    return new StreamExecutionTimer(this, stream, loc);
  }
  void Cuda::CudaContext::tock(Cuda::CudaContext::StreamExecutionTimer *timer,
                               const source_location &loc) {
    // checkError(launchCallback(timer->stream, (void *)recycle_timer, (void *)timer), loc);
    cudri::launchHostFunc(timer->stream, (void *)recycle_timer, (void *)timer);
  }

  void Cuda::CudaContext::setContext(const source_location &loc) const {
    // checkError(hipSetDevice(devid), loc);
    cudri::setContext{getContext()};
  }

  namespace cudri {

    void load_cuda_driver_apis() {
      auto initializer = []() -> bool {
        hipInit(0);
        // int trialVer = CUDA_VERSION - CUDA_VERSION % 1000;
        int trialVer = 11030;
        while (trialVer > 0
               && hipGetProcAddress("hipDrvGetErrorName", (void **)&get_cu_error_name, trialVer,
                                   hipEnableDefault)
                      == hipErrorNotFound)
          trialVer -= 1000;
        const int ver = trialVer;
        fmt::print("final decided driver version: {}\n", ver);
        getchar();
        hipGetProcAddress("hipDrvGetErrorString", (void **)&get_cu_error_string, ver,
                         hipEnableDefault);

#define PER_CUDA_FUNCTION(name, symbol_name, ...)                        \
  hipGetProcAddress(#symbol_name, (void **)&name<__VA_ARGS__>::func, ver, \
                   hipEnableDefault);
#include "cuda_driver_functions.inc.h"
#undef PER_CUDA_FUNCTION
        return true;
      };
      static bool initialized = initializer();
    }
    // template struct name<__VA_ARGS__>;
  }  // namespace cudri

  Cuda::Cuda() {
    fmt::print("[Init -- Begin] Cuda\n");
    cudri::load_cuda_driver_apis();

    numTotalDevice = 0;
    cudri::getDeviceCount{&numTotalDevice};
    contexts.resize(numTotalDevice);
    if (numTotalDevice == 0)
      fmt::print(
          "\t[InitInfo -- DevNum] There are no available device(s) that "
          "support CUDA\n");
    else
      fmt::print("\t[InitInfo -- DevNum] Detected {} CUDA Capable device(s)\n", numTotalDevice);

    for (int i = 0; i < numTotalDevice; i++) {
      auto &context = contexts[i];
      int dev{};
      {
        void *c{nullptr};
        // checkError(hipSetDevice(i), i);
        cudri::getDevice(&dev, i);
        // fmt::print("device ordinal {} is {}\n", i, dev);

        // getContext(&c);
        // cudri::retainDevicePrimaryCtx(&c, dev);
        cudri::createContext(&c, 0,
                             dev);  // hipDeviceScheduleBlockingSync(0x04) | hipDeviceScheduleSpin(0x01)
        context = CudaContext{i, dev, c};
      }

      context.streams.resize((int)StreamIndex::Total);
      for (auto &stream : context.streams)
        // checkError(hipStreamCreateWithFlags((hipStream_t *)&stream, hipStreamNonBlocking),
        // i);
        cudri::createStream(&stream, 0x01);
      context.events.resize((int)EventIndex::Total);
      for (auto &event : context.events)
        // checkError(hipEventCreateWithFlags((hipEvent_t *)&event, hipEventBlockingSync), i);
        cudri::createEvent(&event, 0x01);

      {  ///< device properties
        using namespace cudri;
        int major, minor, multiGpuBoardGroupID, multiProcessorCount, regsPerBlock;
        int supportUnifiedAddressing, supportUm, supportConcurrentUmAccess;
        getDeviceAttribute(&regsPerBlock, (unsigned)hipDeviceAttributeMaxRegistersPerBlock,
                           dev);
        getDeviceAttribute(&multiProcessorCount, (unsigned)hipDeviceAttributeMultiprocessorCount,
                           dev);
        getDeviceAttribute(&multiGpuBoardGroupID,
                           (unsigned)hipDeviceAttributeMultiGpuBoardGroupId, dev);
        getDeviceAttribute(&textureAlignment, (unsigned)hipDeviceAttributeTextureAlignment, dev);
        getDeviceAttribute(&minor, (unsigned)hipDeviceAttributeComputeCapabilityMinor, dev);
        getDeviceAttribute(&major, (unsigned)hipDeviceAttributeComputeCapabilityMajor, dev);
        getDeviceAttribute(&supportUnifiedAddressing,
                           (unsigned)hipDeviceAttributeUnifiedAddressing, dev);
        getDeviceAttribute(&supportUm, (unsigned)hipDeviceAttributeManagedMemory, dev);
        getDeviceAttribute(&supportConcurrentUmAccess,
                           (unsigned)hipDeviceAttributeConcurrentManagedAccess, dev);
        getDeviceAttribute(&context.regsPerMultiprocessor,
                           (unsigned)hipDeviceAttributeMaxRegistersPerMultiprocessor, dev);
        getDeviceAttribute(&context.sharedMemPerMultiprocessor,
                           (unsigned)hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, dev);
        getDeviceAttribute(&context.maxBlocksPerMultiprocessor,
                           (unsigned)hipDeviceAttributeMaxBlocksPerMultiprocessor, dev);
        getDeviceAttribute(&context.sharedMemPerBlock,
                           (unsigned)hipDeviceAttributeMaxSharedMemoryPerBlock, dev);
        getDeviceAttribute(&context.maxThreadsPerBlock,
                           (unsigned)hipDeviceAttributeMaxThreadsPerBlock, dev);
        getDeviceAttribute(&context.maxThreadsPerMultiprocessor,
                           (unsigned)hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);

        context.supportConcurrentUmAccess = supportConcurrentUmAccess;

        fmt::print(
            "\t[InitInfo -- Dev Property] GPU device {} ({}-th group on "
            "board)\n\t\tshared memory per block: {} bytes,\n\t\tregisters per SM: "
            "{},\n\t\tMulti-Processor count: {},\n\t\tSM compute capabilities: "
            "{}.{}.\n\t\tTexture alignment: {} bytes\n\t\tUVM support: allocation({}), unified "
            "addressing({}), concurrent access({})\n",
            i, multiGpuBoardGroupID, context.sharedMemPerBlock, regsPerBlock, multiProcessorCount,
            major, minor, textureAlignment, supportUm, supportUnifiedAddressing,
            supportConcurrentUmAccess);
      }
    }

    /// enable peer access if feasible
    for (int i = 0; i < numTotalDevice; i++) {
      // setContext(contexts[i].getContext());
      // checkError(hipSetDevice(i), i);
      cudri::setContext(contexts[i].getContext());
      for (int j = 0; j < numTotalDevice; j++) {
        if (i != j) {
          int iCanAccessPeer = 0;
          cudri::canAccessPeer(&iCanAccessPeer, contexts[i].getDevice(), contexts[j].getDevice());
          if (iCanAccessPeer) cudri::enablePeerAccess(contexts[j].getContext(), 0);
          fmt::print("\t[InitInfo -- Peer Access] Peer access status {} -> {}: {}\n", i, j,
                     iCanAccessPeer ? "Inactive" : "Active");
        }
      }
    }
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#um-requirements
    /* GPUs with SM architecture 6.x or higher (Pascal class or newer) provide additional
    Unified Memory features such as on-demand page migration and GPU memory oversubscription
    that are outlined throughout this document. Note that currently these features are only
    supported on Linux operating systems. Applications running on Windows (whether in TCC
    or WDDM mode) will use the basic Unified Memory model as on pre-6.x architectures even
    when they are running on hardware with compute capability 6.x or higher. */

    fmt::print("\n[Init -- End] == Finished \'Cuda\' initialization\n\n");
  }

  Cuda::~Cuda() {
    for (int i = 0; i < numTotalDevice; i++) {
      auto &context = contexts[i];
      context.setContext();
      // checkError(hipDeviceSynchronize(), i);
      for (auto stream : context.streams)
        cudri::destroyStream{stream};  // checkError(hipStreamDestroy((hipStream_t)stream), i);
      for (auto event : context.events)
        cudri::destroyEvent{event};  // checkError(hipEventDestroy((hipEvent_t)event), i);
      context.deviceMem.reset(nullptr);
      context.unifiedMem.reset(nullptr);

      // destroyContext(context.getContext());
      // checkError(hipDeviceReset(), i);
      cudri::destroyContext(context.getContext());
    }
    fmt::print("  Finished \'Cuda\' termination\n");
  }

  void Cuda::CudaContext::initDeviceMemory() {
    /// memory
    std::size_t free_byte, total_byte;
    checkError(hipMemGetInfo(&free_byte, &total_byte));
    deviceMem = std::make_unique<MonotonicAllocator>(free_byte >> MEM_POOL_CTRL,
                                                     driver().textureAlignment);
    fmt::print(
        "\t[InitInfo -- memory] device {}\n\t\tfree bytes/total bytes: "
        "{}/{},\n\t\tpre-allocated device memory: {} bytes\n\n",
        getDevId(), free_byte, total_byte, (free_byte >> MEM_POOL_CTRL));
  }
  void Cuda::CudaContext::initUnifiedMemory() {
#if defined(_WIN32)
    throw std::runtime_error("unified virtual memory manually disabled on windows!");
    return;
#endif
    std::size_t free_byte, total_byte;
    checkError(hipMemGetInfo(&free_byte, &total_byte));
    unifiedMem = std::make_unique<MonotonicVirtualAllocator>(getDevId(), total_byte * 4,
                                                             driver().textureAlignment);
    fmt::print(
        "\t[InitInfo -- memory] device {}\n\t\tfree bytes/total bytes: "
        "{}/{},\n\t\tpre-allocated unified memory: {} bytes\n\n",
        getDevId(), free_byte, total_byte, total_byte * 4);
  }

  auto Cuda::CudaContext::borrow(std::size_t bytes) -> void * {
    if (!deviceMem) initDeviceMemory();
    return deviceMem->borrow(bytes);
  }
  void Cuda::CudaContext::resetMem() {
    if (!deviceMem) initDeviceMemory();
    deviceMem->reset();
  }

  auto Cuda::CudaContext::borrowVirtual(std::size_t bytes) -> void * {
#if defined(_WIN32)
    throw std::runtime_error("unified virtual memory manually disabled on windows!");
    return nullptr;
#endif
    if (!unifiedMem) initUnifiedMemory();
    return unifiedMem->borrow(bytes);
  }
  void Cuda::CudaContext::resetVirtualMem() {
    if (!unifiedMem) initUnifiedMemory();
    unifiedMem->reset();
  }

  /// reference: kokkos/core/src/Cuda/Kokkos_Cuda_BlockSize_Deduction.hpp, Ln 101
  int Cuda::deduce_block_size(const Cuda::CudaContext &ctx, void *kernelFunc,
                              std::function<std::size_t(int)> block_size_to_dynamic_shmem,
                              std::string_view kernelName) {
    if (auto it = ctx.funcLaunchConfigs.find(kernelFunc); it != ctx.funcLaunchConfigs.end())
      return it->second.optBlockSize;
    hipFuncAttributes funcAttribs;
    ctx.checkError(hipFuncGetAttributes(&funcAttribs, reinterpret_cast<const void*>(kernelFunc)));
    int optBlockSize{0};

    auto cuda_max_active_blocks_per_sm = [&](int block_size, int dynamic_shmem) {
      // Limits due do registers/SM
      int const regs_per_sm = ctx.regsPerMultiprocessor;
      int const regs_per_thread = funcAttribs.numRegs;
      int const max_blocks_regs = regs_per_sm / (regs_per_thread * block_size);

      // Limits due to shared memory/SM
      size_t const shmem_per_sm = ctx.sharedMemPerMultiprocessor;
      size_t const shmem_per_block = ctx.sharedMemPerBlock;
      size_t const static_shmem = funcAttribs.sharedSizeBytes;
      size_t const dynamic_shmem_per_block = funcAttribs.maxDynamicSharedSizeBytes;
      size_t const total_shmem = static_shmem + dynamic_shmem;

      int const max_blocks_shmem
          = total_shmem > shmem_per_block || dynamic_shmem > dynamic_shmem_per_block
                ? 0
                : (total_shmem > 0 ? (int)shmem_per_sm / total_shmem : max_blocks_regs);

      // Limits due to blocks/SM
      int const max_blocks_per_sm = ctx.maxBlocksPerMultiprocessor;

      // Overall occupancy in blocks
      return std::min({max_blocks_regs, max_blocks_shmem, max_blocks_per_sm});
    };
    auto deduce_opt_block_size = [&]() {
      // Limits
      int const max_threads_per_sm = ctx.maxThreadsPerMultiprocessor;
      // unsure if I need to do that or if this is already accounted for in the functor attributes
      int const min_blocks_per_sm = 1;
      int const max_threads_per_block
          = std::min((int)ctx.maxThreadsPerBlock, funcAttribs.maxThreadsPerBlock);

      // Recorded maximum
      int opt_block_size = 0;
      int opt_threads_per_sm = 0;

      /// iterate all optional blocksize setup
      for (int block_size = max_threads_per_block; block_size > 0; block_size -= 32) {
        size_t const dynamic_shmem = block_size_to_dynamic_shmem(block_size);

        int blocks_per_sm = cuda_max_active_blocks_per_sm(block_size, dynamic_shmem);

        int threads_per_sm = blocks_per_sm * block_size;

        if (threads_per_sm > max_threads_per_sm) {
          blocks_per_sm = max_threads_per_sm / block_size;
          threads_per_sm = blocks_per_sm * block_size;
        }

        // update if higher occupancy (more threads per streaming multiprocessor)
        if (blocks_per_sm >= min_blocks_per_sm) {
          if (threads_per_sm >= opt_threads_per_sm) {
            opt_block_size = block_size;
            opt_threads_per_sm = threads_per_sm;
          }
        }

        // fmt::print("current blocks_sm: {}, threads_sm: {}, size: {}\n", blocks_per_sm,
        //           threads_per_sm, block_size);
        // if (blocks_per_sm != 0) break; // this enabled when querying for maximum block size
      }
      return opt_block_size;
    };

    optBlockSize = deduce_opt_block_size();
    fmt::print(
        fg(fmt::color::lime_green) | fmt::emphasis::bold,
        "{:=^60}\nnumRegs: {}\t\tmaxThreadsPerBlock: {}\nsharedSizeBytes: {}\t"
        "maxDynamicSharedSizeBytes: {}.\n",
        fmt::format(" cuda kernel [{}] optBlockSize [{}] ",
                    kernelName.empty() ? std::to_string((std::uintptr_t)kernelFunc) : kernelName,
                    optBlockSize),
        funcAttribs.numRegs, funcAttribs.maxThreadsPerBlock, funcAttribs.sharedSizeBytes,
        funcAttribs.maxDynamicSharedSizeBytes);
    ctx.funcLaunchConfigs.emplace(kernelFunc, typename Cuda::CudaContext::Config{optBlockSize});
    return optBlockSize;
  }

}  // namespace zs
