#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <utility>

#include "../Logger.hpp"
#include "../Platform.hpp"
#include "Cuda.h"
#include "zensim/cuda/memory/MemOps.hpp"
#include "zensim/execution/ConcurrencyPrimitive.hpp"
#include "zensim/types/SourceLocation.hpp"
#include "zensim/zpc_tpls/fmt/color.h"
#include "zensim/zpc_tpls/fmt/format.h"

#define MEM_POOL_CTRL 3

#if 0
namespace {
  static zs::Mutex g_cudaMutex;
  static std::atomic<bool> g_isCudaInitialized = false;
  static zs::Cuda *g_cudaInstance = nullptr;
}  // namespace
#endif

namespace zs {

#if 0
  Cuda &Cuda::instance() {
    if (g_isCudaInitialized.load(std::memory_order_acquire)) return *g_cudaInstance;
    g_cudaMutex.lock();
    if (g_isCudaInitialized.load(std::memory_order_acquire)) return *g_cudaInstance;

    if (!g_cudaInstance) g_cudaInstance = new Cuda;

    g_isCudaInitialized.store(true, std::memory_order_release);
    g_cudaMutex.unlock();
    return *g_cudaInstance;
  }
#endif

  Cuda::ContextGuard::ContextGuard(void *context, bool restore, const source_location &loc)
      : needRestore(false), loc(loc) {
    if (context) {
      if (restore)
        if (checkCuApiError(hipCtxGetCurrent((hipCtx_t *)(&prevContext)), loc,
                            "[hipCtxGetCurrent]")) {
          if (context != prevContext)
            needRestore
                = checkCuApiError(hipCtxSetCurrent((hipCtx_t)context), loc, "[hipCtxGetCurrent]");
        }
    }
  }
  Cuda::ContextGuard::~ContextGuard() {
    if (needRestore)
      if (hipError_t ec = hipCtxSetCurrent((hipCtx_t)prevContext); ec != hipSuccess) {
        const char *errString = nullptr;
        if (hipDrvGetErrorString) {
          hipDrvGetErrorString(ec, &errString);
          checkCuApiError((u32)ec, loc, fmt::format("on restoring context {}", prevContext),
                          errString);
        } else
          checkCuApiError((u32)ec, loc, fmt::format("on restoring context {}", prevContext));
      }
  }
  /*
    __device__ __constant__ char g_cuda_constant_cache[8192];  // 1024 words

    void Cuda::init_constant_cache(void *ptr, size_t size) {
      hipMemcpyToSymbol(HIP_SYMBOL(g_cuda_constant_cache), ptr, size, 0, hipMemcpyHostToDevice);
      // cudri::memcpyHtoD((void *)g_cuda_constant_cache, ptr, size);
    }
    void Cuda::init_constant_cache(void *ptr, size_t size, void *stream) {
      hipMemcpyToSymbolAsync(HIP_SYMBOL(g_cuda_constant_cache), ptr, size, 0, hipMemcpyHostToDevice,
                              (hipStream_t)stream);
    }
    */

  /// error handling
  u32 Cuda::get_last_cuda_rt_error() { return (u32)hipPeekAtLastError(); }

  std::string_view Cuda::get_cuda_rt_error_string(u32 errorCode) {
    // return hipGetErrorString((hipError_t)errorCode);
    return hipGetErrorString((hipError_t)errorCode);
  }
  void Cuda::check_cuda_rt_error(u32 errorCode, ProcID did, const source_location &loc) {
    if (errorCode != 0) {
      if (did >= 0) {
        auto &context = Cuda::context(did);
        if (context.errorStatus) return;  // there already exists a preceding cuda error
        context.errorStatus = true;
      }
      const auto fileInfo = fmt::format("# File: \"{:<50}\"", loc.file_name());
      const auto locInfo = fmt::format("# Ln {}, Col {}", loc.line(), loc.column());
      const auto funcInfo = fmt::format("# Func: \"{}\"", loc.function_name());
#if 0
      fmt::print(fg(fmt::color::crimson) | fmt::emphasis::italic | fmt::emphasis::bold,
                 "\nCuda Error on Device {}: {}\n{:=^60}\n{}\n{}\n{}\n{:=^60}\n\n",
                 did >= 0 ? std::to_string(did) : "unknown", get_cuda_rt_error_string(errorCode),
                 " cuda api error location ", fileInfo, locInfo, funcInfo, "=");
#else
      std::cerr << fmt::format("\nCuda Error on Device {}: {}\n{:=^60}\n{}\n{}\n{}\n{:=^60}\n\n",
                               did >= 0 ? std::to_string(did) : "unknown",
                               get_cuda_rt_error_string(errorCode), " cuda api error location ",
                               fileInfo, locInfo, funcInfo, "=");
#endif
    }
  }

  /// kernel launch
  u32 Cuda::launchKernel(const void *f, unsigned int gx, unsigned int gy, unsigned int gz,
                         unsigned int bx, unsigned int by, unsigned int bz, void **args,
                         size_t shmem, void *stream) {
    return hipLaunchKernel(reinterpret_cast<const void*>(f), dim3{gx, gy, gz}, dim3{bx, by, bz}, args, shmem,
                            (hipStream_t)stream);
    // return cudri::launchCuKernel(const_cast<void *>(f), gx, gy, gz, bx, by, bz, (unsigned
    // int)shmem,
    //                      stream, args, (void **)nullptr);
  }
  u32 Cuda::launchCooperativeKernel(const void *f, unsigned int gx, unsigned int gy,
                                    unsigned int gz, unsigned int bx, unsigned int by,
                                    unsigned int bz, void **args, size_t shmem, void *stream) {
    // return cudri::launchCuCooperativeKernel(const_cast<void *>(f), gx, gy, gz, bx, by, bz, shmem,
    // stream, args);
    return hipLaunchCooperativeKernel(reinterpret_cast<const void*>(f), dim3{gx, gy, gz}, dim3{bx, by, bz}, args, shmem,
                                       (hipStream_t)stream);
  }
  u32 Cuda::launchCallback(void *stream, void *f, void *data) {
    // return hipLaunchHostFunc((hipStream_t)stream, (hipHostFn_t)f, data);
    return (u32)hipLaunchHostFunc((hipStream_t)stream, (hipHostFn_t)f, data);
  }

  void Cuda::CudaContext::checkError(u32 errorCode, const source_location &loc) const {
    /// only shows the first error message
    Cuda::check_cuda_rt_error(errorCode, getDevId(), loc);
  }

  // record
  void Cuda::CudaContext::recordEventCompute(const source_location &loc) {
    checkError(hipEventRecord((hipEvent_t)eventCompute(), (hipStream_t)streamCompute()), loc);
    // hipEventRecord((hipEvent_t)eventCompute(), (hipStream_t)streamCompute());
  }
  void Cuda::CudaContext::recordEventSpare(StreamID id, const source_location &loc) {
    checkError(hipEventRecord((hipEvent_t)eventSpare(id), (hipStream_t)streamSpare(id)), loc);
    // hipEventRecord((hipEvent_t)eventSpare(id), (hipStream_t)streamSpare(id));
  }
  // sync
  void Cuda::CudaContext::syncStream(StreamID sid, const source_location &loc) const {
    checkError(hipStreamSynchronize((hipStream_t)stream(sid)), loc);
    // hipStreamSynchronize((hipStream_t)stream(sid));
  }
  void Cuda::CudaContext::syncCompute(const source_location &loc) const {
    checkError(hipStreamSynchronize((hipStream_t)streamCompute()), loc);
    // hipStreamSynchronize((hipStream_t)streamCompute());
  }
  void Cuda::CudaContext::syncStreamSpare(StreamID sid, const source_location &loc) const {
    checkError(hipStreamSynchronize((hipStream_t)streamSpare(sid)), loc);
    // hipStreamSynchronize((hipStream_t)streamSpare(sid));
  }
  // stream-event sync
  void Cuda::CudaContext::computeStreamWaitForEvent(void *event, const source_location &loc) {
    checkError(hipStreamWaitEvent((hipStream_t)streamCompute(), (hipEvent_t)event, 0), loc);
    // hipStreamWaitEvent((hipStream_t)streamCompute(), (hipEvent_t)event, 0);
  }
  void Cuda::CudaContext::spareStreamWaitForEvent(StreamID sid, void *event,
                                                  const source_location &loc) {
    checkError(hipStreamWaitEvent((hipStream_t)streamSpare(sid), (hipEvent_t)event, 0), loc);
    // hipStreamWaitEvent((hipStream_t)streamSpare(sid), (hipEvent_t)event, 0);
  }
  void *Cuda::CudaContext::streamMemAlloc(size_t size, void *stream, const source_location &loc) {
    void *ptr;
    hipMallocAsync((hipDeviceptr_t *)&ptr, size, (hipStream_t)stream);
    return ptr;
  }
  void Cuda::CudaContext::streamMemFree(void *ptr, void *stream, const source_location &loc) {
    hipFreeAsync((hipDeviceptr_t)ptr, (hipStream_t)stream);
  }
  Cuda::CudaContext::StreamExecutionTimer *Cuda::CudaContext::tick(void *stream,
                                                                   const source_location &loc) {
    return new StreamExecutionTimer(this, stream, loc);
  }
  void Cuda::CudaContext::tock(Cuda::CudaContext::StreamExecutionTimer *timer,
                               const source_location &loc) {
    hipLaunchHostFunc((hipStream_t)timer->stream, (hipHostFn_t)recycle_timer, (void *)timer);
  }

  void Cuda::CudaContext::setContext(const source_location &loc) const {
    const char *errString = nullptr;
    auto ec = hipCtxSetCurrent((hipCtx_t)getContext());
    if (ec != hipSuccess) {
      hipDrvGetErrorString(ec, &errString);
      checkCuApiError((u32)ec, loc, "[Cuda::CudaContext::setContext]", errString);
    }
  }

  bool Cuda::set_default_device(int dev, const source_location &loc) {
    auto &inst = driver();
    if (dev == inst.defaultDevice || dev >= inst.numTotalDevice || dev < 0) return false;
    inst.defaultDevice = dev;
    return prepare_context(mem_device, dev, loc);
  }
  int Cuda::get_default_device() noexcept { return driver().defaultDevice; }

  Cuda::Cuda() {
    fmt::print("[Init -- Begin] Cuda\n");
    errorStatus = false;
    hipError_t res = hipInit(0);

    numTotalDevice = 0;
    hipGetDeviceCount(&numTotalDevice);
    contexts.resize(numTotalDevice);
    if (numTotalDevice == 0)
      fmt::print(
          "\t[InitInfo -- DevNum] There are no available device(s) that "
          "support CUDA\n");
    else
      fmt::print("\t[InitInfo -- DevNum] Detected {} CUDA Capable device(s)\n", numTotalDevice);

    defaultDevice = 0;
    {
      hipCtx_t ctx = nullptr;
      auto ec = hipCtxGetCurrent(&ctx);
      if (ec != hipSuccess) {
        const char *errString = nullptr;
        hipDrvGetErrorString(ec, &errString);
        checkCuApiError((u32)ec, errString);
      } else {
        int devid = defaultDevice;
        if (ctx != NULL) {
          auto ec = hipCtxGetDevice(&devid);
          if (ec != hipSuccess) {
            const char *errString = nullptr;
            hipDrvGetErrorString(ec, &errString);
            checkCuApiError((u32)ec, errString);
          } else
            defaultDevice = devid;  // record for restore later
        }                           // otherwise, no context has been initialized yet.
      }
    }
    for (int i = 0; i < numTotalDevice; i++) {
      auto &context = contexts[i];
      int dev{};
      {
        void *ctx{nullptr};
        // checkError(hipSetDevice(i), i);
        hipDeviceGet((hipDevice_t *)&dev, i);
        fmt::print("device ordinal {} has handle {}\n", i, dev);

        unsigned int ctxFlags, expectedFlags = hipDeviceScheduleAuto;
        // unsigned int ctxFlags, expectedFlags = hipDeviceScheduleBlockingSync;
        int isActive;
        hipDevicePrimaryCtxGetState((hipDevice_t)dev, &ctxFlags, &isActive);

        /// follow tensorflow's impl
        if (ctxFlags != expectedFlags) {
          if (isActive) {
            ZS_ERROR(
                fmt::format("The primary active context has flag [{}], but [{}] is expected.\n",
                            ctxFlags, expectedFlags)
                    .data());
          } else {
            hipDevicePrimaryCtxSetFlags((hipDevice_t)dev, expectedFlags);
          }
        }

        void *formerCtx;
        int formerDev;
        hipCtxGetCurrent((hipCtx_t *)&formerCtx);
        res = hipDevicePrimaryCtxRetain((hipCtx_t *)&ctx, (hipDevice_t)dev);
        if (formerCtx != nullptr) {
          hipCtxGetDevice(&formerDev);
          ZS_ERROR_IF(formerDev == dev,
                      fmt::format("setting device [{}], yet the current device handle is {}.", dev,
                                  formerDev));
          if (formerCtx == ctx) {
            ZS_INFO(fmt::format("The primary context [{}] for device {} exists.", formerCtx,
                                formerDev));
          } else {
            ZS_WARN(fmt::format(
                "A non-primary context [{}] for device {} exists. The primary context is now {}.",
                formerCtx, formerDev, ctx));
          }
        }
        hipCtxSetCurrent((hipCtx_t)ctx);  // not sure why this is meaningful
        if (res == hipSuccess) {
          // add this new context
          context = CudaContext{i, dev, ctx};
        } else if (res == hipErrorOutOfMemory) {
          size_t nbs;
          hipDeviceTotalMem(&nbs, (hipDevice_t)dev);
          ZS_WARN(fmt::format("{} bytes in total for device {}.", nbs, dev));
        }
      }

      context.streams.resize((int)StreamIndex::Total);
      for (auto &stream : context.streams)
        hipStreamCreateWithFlags((hipStream_t *)&stream, hipStreamDefault);  // safer to sync with stream 0
      /// @note event for default stream is the last
      context.events.resize((int)EventIndex::Total);
      for (auto &event : context.events) hipEventCreateWithFlags((hipEvent_t *)&event, hipEventBlockingSync);

      {  ///< device properties
        int major, minor, multiGpuBoardGroupID, regsPerBlock;
        int supportUnifiedAddressing, supportUm, supportConcurrentUmAccess;
        hipDeviceGetAttribute(&regsPerBlock, hipDeviceAttributeMaxRegistersPerBlock, dev);
        hipDeviceGetAttribute(&multiGpuBoardGroupID, hipDeviceAttributeMultiGpuBoardGroupId,
                             (hipDevice_t)dev);
        hipDeviceGetAttribute(&textureAlignment, hipDeviceAttributeTextureAlignment, dev);
        hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, (hipDevice_t)dev);
        hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, (hipDevice_t)dev);
        hipDeviceGetAttribute(&supportUnifiedAddressing, hipDeviceAttributeUnifiedAddressing,
                             (hipDevice_t)dev);
        hipDeviceGetAttribute(&supportUm, hipDeviceAttributeManagedMemory, (hipDevice_t)dev);
        hipDeviceGetAttribute(&supportConcurrentUmAccess,
                             hipDeviceAttributeConcurrentManagedAccess, (hipDevice_t)dev);
        hipDeviceGetAttribute(&context.numMultiprocessor, hipDeviceAttributeMultiprocessorCount,
                             (hipDevice_t)dev);
        hipDeviceGetAttribute(&context.regsPerMultiprocessor,
                             hipDeviceAttributeMaxRegistersPerMultiprocessor, (hipDevice_t)dev);
        hipDeviceGetAttribute(&context.sharedMemPerMultiprocessor,
                             hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, dev);
        hipDeviceGetAttribute(&context.maxBlocksPerMultiprocessor,
                             hipDeviceAttributeMaxBlocksPerMultiprocessor, (hipDevice_t)dev);
        hipDeviceGetAttribute(&context.sharedMemPerBlock,
                             hipDeviceAttributeMaxSharedMemoryPerBlock, (hipDevice_t)dev);
        hipDeviceGetAttribute(&context.maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock,
                             (hipDevice_t)dev);
        hipDeviceGetAttribute(&context.maxThreadsPerMultiprocessor,
                             hipDeviceAttributeMaxThreadsPerMultiProcessor, (hipDevice_t)dev);

        context.supportConcurrentUmAccess = supportConcurrentUmAccess;

        fmt::print(
            "\t[InitInfo -- Dev Property] GPU device {} ({}-th group on "
            "board)\n\t\tshared memory per block: {} bytes,\n\t\tregisters per SM: "
            "{},\n\t\tMulti-Processor count: {},\n\t\tSM compute capabilities: "
            "{}.{}.\n\t\tTexture alignment: {} bytes\n\t\tUVM support: allocation({}), unified "
            "addressing({}), concurrent access({})\n",
            i, multiGpuBoardGroupID, context.sharedMemPerBlock, regsPerBlock,
            context.numMultiprocessor, major, minor, textureAlignment, supportUm,
            supportUnifiedAddressing, supportConcurrentUmAccess);
      }
    }

    /// enable peer access if feasible
    for (int i = 0; i < numTotalDevice; i++) {
      // checkError(hipSetDevice(i), i);
      hipCtxSetCurrent((hipCtx_t)contexts[i].getContext());
      for (int j = 0; j < numTotalDevice; j++) {
        if (i != j) {
          int iCanAccessPeer = 0;
          hipDeviceCanAccessPeer(&iCanAccessPeer, contexts[i].getDevice(), contexts[j].getDevice());
          if (iCanAccessPeer) hipCtxEnablePeerAccess((hipCtx_t)contexts[j].getContext(), 0);
          fmt::print("\t[InitInfo -- Peer Access] Peer access status {} -> {}: {}\n", i, j,
                     iCanAccessPeer ? "Inactive" : "Active");
        }
      }
    }
    // select gpu 0 by default
    hipCtxSetCurrent((hipCtx_t)contexts[defaultDevice].getContext());
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#um-requirements
    /* GPUs with SM architecture 6.x or higher (Pascal class or newer) provide additional
    Unified Memory features such as on-demand page migration and GPU memory oversubscription
    that are outlined throughout this document. Note that currently these features are only
    supported on Linux operating systems. Applications running on Windows (whether in TCC
    or WDDM mode) will use the basic Unified Memory model as on pre-6.x architectures even
    when they are running on hardware with compute capability 6.x or higher. */

    fmt::print("\n[Init -- End] == Finished \'Cuda\' initialization\n\n");
  }

  Cuda::~Cuda() {
    // do not do anything, let driver recycle resources
  }

  /// reference: kokkos/core/src/Cuda/Kokkos_Cuda_BlockSize_Deduction.hpp, Ln 101
  int Cuda::deduce_block_size(const source_location &loc, const Cuda::CudaContext &ctx,
                              void *kernelFunc, function<size_t(int)> block_size_to_dynamic_shmem,
                              std::string_view kernelName) {
    if (auto it = ctx.funcLaunchConfigs.find(kernelFunc); it != ctx.funcLaunchConfigs.end())
      return it->second.optBlockSize;
    hipFuncAttributes funcAttribs;
    ctx.checkError(hipFuncGetAttributes(&funcAttribs, reinterpret_cast<const void*>(kernelFunc)), loc);
    int optBlockSize{0};

    auto cuda_max_active_blocks_per_sm = [&](int block_size, int dynamic_shmem) {
      // Limits due do registers/SM
      int const regs_per_sm = ctx.regsPerMultiprocessor;
      int const regs_per_thread = funcAttribs.numRegs;
      int const max_blocks_regs = regs_per_sm / (regs_per_thread * block_size);

      // Limits due to shared memory/SM
      size_t const shmem_per_sm = ctx.sharedMemPerMultiprocessor;
      size_t const shmem_per_block = ctx.sharedMemPerBlock;
      size_t const static_shmem = funcAttribs.sharedSizeBytes;
      size_t const dynamic_shmem_per_block = funcAttribs.maxDynamicSharedSizeBytes;
      size_t const total_shmem = static_shmem + dynamic_shmem;

      int const max_blocks_shmem
          = total_shmem > shmem_per_block || dynamic_shmem > dynamic_shmem_per_block
                ? 0
                : (total_shmem > 0 ? (int)shmem_per_sm / total_shmem : max_blocks_regs);

      // Limits due to blocks/SM
      int const max_blocks_per_sm = ctx.maxBlocksPerMultiprocessor;

      // Overall occupancy in blocks
      return std::min({max_blocks_regs, max_blocks_shmem, max_blocks_per_sm});
    };
    auto deduce_opt_block_size = [&]() {
      // Limits
      int const max_threads_per_sm = ctx.maxThreadsPerMultiprocessor;
      // unsure if I need to do that or if this is already accounted for in the functor attributes
      int const min_blocks_per_sm = 1;
      int const max_threads_per_block
          = std::min((int)ctx.maxThreadsPerBlock, funcAttribs.maxThreadsPerBlock);

      // Recorded maximum
      int opt_block_size = 0;
      int opt_threads_per_sm = 0;

      /// iterate all optional blocksize setup
      for (int block_size = max_threads_per_block; block_size > 0; block_size -= 32) {
        size_t const dynamic_shmem = block_size_to_dynamic_shmem(block_size);

        int blocks_per_sm = cuda_max_active_blocks_per_sm(block_size, dynamic_shmem);

        int threads_per_sm = blocks_per_sm * block_size;

        if (threads_per_sm > max_threads_per_sm) {
          blocks_per_sm = max_threads_per_sm / block_size;
          threads_per_sm = blocks_per_sm * block_size;
        }

        // update if higher occupancy (more threads per streaming multiprocessor)
        if (blocks_per_sm >= min_blocks_per_sm) {
          if (threads_per_sm >= opt_threads_per_sm) {
            opt_block_size = block_size;
            opt_threads_per_sm = threads_per_sm;
          }
        }

        // fmt::print("current blocks_sm: {}, threads_sm: {}, size: {}\n", blocks_per_sm,
        //           threads_per_sm, block_size);
        // if (blocks_per_sm != 0) break; // this enabled when querying for maximum block size
      }
      return opt_block_size;
    };

    optBlockSize = deduce_opt_block_size();
    fmt::print(
        fg(fmt::color::lime_green) | fmt::emphasis::bold,
        "{:=^60}\nnumRegs: {}\t\tmaxThreadsPerBlock: {}\nsharedSizeBytes: {}\t"
        "maxDynamicSharedSizeBytes: {}.\n",
        fmt::format(" cuda kernel [{}] optBlockSize [{}] ",
                    kernelName.empty() ? std::to_string((std::uintptr_t)kernelFunc) : kernelName,
                    optBlockSize),
        funcAttribs.numRegs, funcAttribs.maxThreadsPerBlock, funcAttribs.sharedSizeBytes,
        funcAttribs.maxDynamicSharedSizeBytes);
    ctx.funcLaunchConfigs.emplace(kernelFunc, typename Cuda::CudaContext::Config{optBlockSize});
    return optBlockSize;
  }

}  // namespace zs
